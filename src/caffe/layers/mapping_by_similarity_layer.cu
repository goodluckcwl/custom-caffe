#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/mapping_by_similarity_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void mapping_forward_gpu_kernel(const int n_kernels, const Dtype* data_im1, const Dtype* data_im2,
                                        const int channels, const int height, const int width,
                                        const Dtype* data_sim, Dtype alpha,
                                           Dtype* output_im1, Dtype* output_im2){
    CUDA_KERNEL_LOOP(index, n_kernels) {
        const int w = index % width;
        const int h = index / width;

        const Dtype* data_im1_ptr = data_im1;
        data_im1_ptr += h * width + w;
        const Dtype* data_im2_ptr = data_im2;
        data_im2_ptr += h * width + w;
        Dtype* output_im1_ptr = output_im1;
        output_im1_ptr += h * width + w;
        Dtype* output_im2_ptr = output_im2;
        output_im2_ptr += h * width + w;

        Dtype beta = *(data_sim + h * width + w);
        const int channel_size = width * height;
        for (int c = 0; c < channels; ++c) {
            Dtype x1 = *(data_im1_ptr + c * channel_size);
            Dtype x2 = *(data_im2_ptr + c * channel_size);
            output_im1_ptr[c * channel_size] = (x1 + alpha*(1-beta) * x2)/(1 + alpha - alpha*beta);
            output_im2_ptr[c * channel_size] = (x2 + alpha*(1-beta) * x1)/(1 + alpha - alpha*beta);

        }
    }
}

template <typename Dtype>
__global__ void image_backward_gpu_kernel(const int n_kernels, const Dtype* top_diff1, const Dtype* top_diff2,
                                        const int channels, const int height, const int width,
                                        const Dtype* data_sim, Dtype alpha,
                                          Dtype* bottom_diff1, Dtype* bottom_diff2){
    CUDA_KERNEL_LOOP(index, n_kernels) {
        const int w = index % width;
        const int h = index / width;

        const Dtype* top_diff1_ptr = top_diff1;
        top_diff1_ptr += h * width + w;
        const Dtype* top_diff2_ptr = top_diff2;
        top_diff2_ptr += h * width + w;

        Dtype* bottom_diff1_ptr = bottom_diff1;
        bottom_diff1_ptr += h * width + w;
        Dtype* bottom_diff2_ptr = bottom_diff2;
        bottom_diff2_ptr += h * width + w;

        const int channel_size = width * height;
        Dtype beta = *(data_sim + h * width + w);
        for (int c = 0; c < channels; ++c) {
            Dtype d1_diff = top_diff1_ptr[c * channel_size];
            Dtype d2_diff = top_diff2_ptr[c * channel_size];
            Dtype factor = 1+alpha-alpha*beta;
            bottom_diff1_ptr[c * channel_size] += 1/factor * d1_diff + alpha*(1-beta)/factor*d2_diff;
            bottom_diff2_ptr[c * channel_size] += 1/factor * d2_diff + alpha*(1-beta)/factor*d1_diff;
        }
    }
}

template <typename Dtype>
__global__ void similarity_backward_gpu_kernel(const int n_kernels, const Dtype* bottom_data1, const Dtype* bottom_data2,
                                            const int channels, const int height, const int width,
                                            const Dtype* top_diff1, const Dtype* top_diff2,
                                            const Dtype* data_sim,
                                               Dtype alpha,
                                               Dtype* diff_sim){
    CUDA_KERNEL_LOOP(index, n_kernels) {
        const int w = index % width;
        const int h = index / width;

        const Dtype* bottom_data1_ptr = bottom_data1;
        bottom_data1_ptr += h * width + w;
        const Dtype* bottom_data2_ptr = bottom_data2;
        bottom_data2_ptr += h * width + w;

        const Dtype* top_diff1_ptr = top_diff1;
        top_diff1_ptr += h * width + w;
        const Dtype* top_diff2_ptr = top_diff2;
        top_diff2_ptr += h * width + w;

        Dtype* diff_sim_ptr = diff_sim;
        diff_sim_ptr += h * width + w;

        const int channel_size = width * height;
        Dtype beta = data_sim[h * width + w];
        for (int c = 0; c < channels; ++c) {
            Dtype x1 = bottom_data1_ptr[c * channel_size];
            Dtype x2 = bottom_data2_ptr[c * channel_size];
            Dtype factor = 1 + alpha - alpha * beta;
            Dtype factor1 =(alpha*x1 - alpha*x2)/(factor*factor);
            Dtype factor2 =(alpha*x2 - alpha*x1)/(factor*factor);
            // Accumulate diffs
            Dtype d1_diff = top_diff1_ptr[c * channel_size];
            Dtype d2_diff = top_diff2_ptr[c * channel_size];
            *diff_sim_ptr += factor1 * d1_diff + factor2 * d2_diff;
        }
    }
}

template <typename Dtype>
void MappingBySimilarityLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    if(false){
        this->Forward_cpu(bottom, top);
    }
    const Dtype *bottom_data = bottom[0]->gpu_data();
    const Dtype *sim_data = bottom[1]->gpu_data();
    Dtype *top_data = top[0]->mutable_gpu_data();
    int n_images = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = bottom[0]->shape(2);
    int width = bottom[0]->shape(3);

    int channel_size = bottom[0]->count(2);
    // We lanch width * height kernels.
    const int num_kernels = width * height;
    for (int n = 0; n < n_images/2; ++n) {
        const Dtype *s1 = bottom_data + 2*n * bottom[0]->count(1);
        const Dtype *s2 = bottom_data + (2*n+1) * bottom[0]->count(1);
        Dtype *d1 = top_data + 2 * n * top[0]->count(1);
        Dtype *d2 = top_data + (2*n+1) * top[0]->count(1);
        mapping_forward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                CAFFE_CUDA_NUM_THREADS>>>(
                        num_kernels, s1, s2, channels, height, width,
                        sim_data + n * bottom[1]->count(1),
                        //alpha
                        alpha_,
                        //Output
                        d1,d2);
    }

}

template <typename Dtype>
void MappingBySimilarityLayer<Dtype>::Backward_gpu(const std::vector<caffe::Blob<Dtype> *> &top,
                                                   const std::vector<bool> &propagate_down,
                                                   const std::vector<caffe::Blob<Dtype> *> &bottom) {
    if (propagate_down[0]) {
        if(false){
            this->Backward_cpu(top, propagate_down, bottom);
        }
        // gradient w.r.t. image. Note that we will accumulate diffs.
        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
        const Dtype *top_diff = top[0]->gpu_diff();
        const Dtype *sim_data = bottom[1]->gpu_data();
        int n_images = bottom[0]->shape(0);
        int channels = bottom[0]->shape(1);
        int height = bottom[0]->shape(2);
        int width = bottom[0]->shape(3);

        // Clear grad
        caffe_gpu_set(bottom[0]->count(0), Dtype(0.0), bottom_diff);
        int channel_size = bottom[0]->count(2);
        // We launch width*height kernels.
        const int num_kernels = width * height;
        for (int n = 0; n < n_images/2; ++n) {
            Dtype *diff1 = bottom_diff + 2*n * bottom[0]->count(1) ;
            Dtype *diff2 = bottom_diff + (2*n+1) * bottom[0]->count(1);
            const Dtype *d1 = top_diff + 2*n * top[0]->count(1) ;
            const Dtype *d2 = top_diff + (2*n+1) * top[0]->count(1);
            image_backward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                    CAFFE_CUDA_NUM_THREADS>>>(
                            num_kernels, d1, d2, channels, height, width,
                                    sim_data + n * bottom[1]->count(1),
                                    alpha_,
                                    //Output
                                    diff1,diff2);
        }
    }
    if (propagate_down[1]) {
        Dtype* sim_diff = bottom[1]->mutable_gpu_diff();
        const Dtype *sim_data = bottom[1]->gpu_data();

        const Dtype* bottom_data = bottom[0]->gpu_data();
        const Dtype *top_diff = top[0]->gpu_diff();

        int n_images = bottom[0]->shape(0);
        int channels = bottom[0]->shape(1);
        int height = bottom[0]->shape(2);
        int width = bottom[0]->shape(3);
        int channel_size = bottom[0]->count(2);

        // Clear grads. Because we need to accumulate diffs.
        // Note that the similarity map has only one channel.
        caffe_gpu_set(bottom[1]->count(0), Dtype(0.0), sim_diff);
        // We launch width*height kernels.
        const int num_kernels = width * height;
        for (int n = 0; n < n_images/2; ++n) {
            const Dtype *s1 = bottom_data + 2*n * bottom[0]->count(1) ;
            const Dtype *s2 = bottom_data + (2*n+1) * bottom[0]->count(1) ;
            Dtype *diff = sim_diff + n * bottom[1]->count(1) ;
            const Dtype *d1 = top_diff + 2*n * top[0]->count(1) ;
            const Dtype *d2 = top_diff + (2*n+1) * top[0]->count(1);
            similarity_backward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                    CAFFE_CUDA_NUM_THREADS>>>(
                            num_kernels, s1, s2, channels, height, width,
                                    d1, d2,
                                    sim_data + n * bottom[1]->count(1),
                                    alpha_,
                                    //Output
                                    diff);
        }

    }
}


INSTANTIATE_LAYER_GPU_FUNCS(MappingBySimilarityLayer);

}  // namespace caffe
